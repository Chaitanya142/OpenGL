
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{

unsigned int x=blockIdx.x*blockDim.x+threadIdx.x;
unsigned int y=blockIdx.y*blockDim.y+threadIdx.y;

float u=x/(float)width;
float v=y/(float)height;

u=u*2.0f-1.0f;
v=v*2.0f-1.0f;

float frequency=4.0f;

float w=sinf(u*frequency+time)*cosf(v*frequency+time)*0.5f;

pos[y*width+x]=make_float4(u,w,v,1.0);
}

extern FILE* gpFile;
void LaunchCUDAKernel(float4 *pos, unsigned int meshWidth, unsigned int meshHeight, float time)
{
	fprintf(gpFile, "\nInside GPU %f", time);
	dim3 block(8,8,1);

	dim3 grid(meshWidth/block.x, meshHeight/block.y,1);
	simple_vbo_kernel<<<grid,block>>>(pos,meshWidth,meshHeight,time);

}

